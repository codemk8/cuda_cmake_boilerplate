#include "hip/hip_runtime.h"
#include <iostream>
#include "my_kernel.h"

using namespace std;
constexpr int con = 3; // C++11 feature

__global__ void my_kernel() {
  printf("inside my kernel con is %d\n", con);
  // call the real kernel <<< >>>
}

void my_kernel_wrapper() {
  my_kernel<<<1, 1>>>();
}
